#include <stdio.h>
#include <stdlib.h>

#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

#include "image.h"

using namespace rfgpu;

Image::Image() {
    plan = 0;
    xpix = ypix = 0;
}

Image::~Image() {
    if (plan) hipfftDestroy(plan);
}

void Image::setup() {
    hipfftResult_t rv;
    rv = hipfftPlan2d(&plan, xpix, ypix, HIPFFT_C2R);
    if (rv != HIPFFT_SUCCESS) {
        char msg[1024];
        sprintf(msg, "Image::setup error planning FFT (%d)", rv);
        throw std::runtime_error(msg);
    }
}

void Image::operate(hipfftComplex *vis, hipfftReal *img) {
    hipfftResult_t rv;
    rv = hipfftExecC2R(plan, vis, img);
    if (rv != HIPFFT_SUCCESS) {
        char msg[1024];
        sprintf(msg, "Image::operate error executing FFT (%d)", rv);
        throw std::runtime_error(msg);
    }
}

