#include <stdio.h>
#include <stdlib.h>

#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

#include "image.h"

using namespace rfgpu;

Image::Image(int _xpix, int _ypix) {
    plan = 0;
    xpix = _xpix;
    ypix = _ypix;
    setup();
}

Image::~Image() {
    if (plan) hipfftDestroy(plan);
}

void Image::setup() {
    hipfftResult_t rv;
    rv = hipfftPlan2d(&plan, xpix, ypix, HIPFFT_C2R);
    if (rv != HIPFFT_SUCCESS) {
        char msg[1024];
        sprintf(msg, "Image::setup error planning FFT (%d)", rv);
        throw std::runtime_error(msg);
    }
}

void Image::operate(Array<cdata,true> &vis, Array<rdata,true> &img) {
    if (vis.len() != vispix()) {
        char msg[1024];
        sprintf(msg, "Image::operate vis array size (%d) != expected (%d)",
                vis.len(), vispix());
        throw std::invalid_argument(msg);
    }
    if (img.len() != imgpix()) {
        char msg[1024];
        sprintf(msg, "Image::operate img array size (%d) != expected (%d)",
                img.len(), imgpix());
        throw std::invalid_argument(msg);
    }
    operate(vis.d, img.d);
}

void Image::operate(cdata *vis, rdata *img) {
    hipfftResult_t rv;
    rv = hipfftExecC2R(plan, vis, img);
    if (rv != HIPFFT_SUCCESS) {
        char msg[1024];
        sprintf(msg, "Image::operate error executing FFT (%d)", rv);
        throw std::runtime_error(msg);
    }
}

