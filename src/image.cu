#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

#include "image.h"

using namespace rfgpu;

Image::Image() {
    plan = NULL; 
    xpix = ypix = 0;
}

Image::~Image() {
    if (plan) hipfftDestroy(plan);
}

void Image::setup() {
    hipfftPlan2d(&plan, xpix, ypix, HIPFFT_C2R); // TODO check for error
}

void Image::operate(hipfftComplex *vis, hipfftReal *img) {
    hipfftExecC2R(plan, vis, img);
}

