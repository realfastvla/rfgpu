
#include <hip/hip_runtime.h>

#include "timer.h"

using namespace rfgpu;

Timer::Timer() {
    n_call = 0;
    t_total = 0.0;
    hipEventCreate(&event0);
    hipEventCreate(&event1);
}

Timer::~Timer() {
    hipEventDestroy(event0);
    hipEventDestroy(event1);
}

void Timer::start() {
    hipEventRecord(event0);
}

void Timer::stop() {
    hipEventRecord(event1);
    hipEventSynchronize(event1);
    float t;
    hipEventElapsedTime(&t, event0, event1);
    t_total += t;
    n_call++;
}

