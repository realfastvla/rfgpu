#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <vector>
#include <algorithm>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hipsparse.h>

#include "grid.h"

using namespace rfgpu;

// TODO raise exception
#define cusparse_check_rv(func) \
    if (rv!=HIPSPARSE_STATUS_SUCCESS) { \
        char msg[1024]; \
        sprintf(msg, "cusparse error: %s returned %d", func, rv); \
        throw std::runtime_error(msg); \
    }

Grid::Grid(int _nbl, int _nchan, int _ntime, int _upix, int _vpix) {
    nbl = _nbl;
    nchan = _nchan;
    ntime = _ntime;
    upix = _upix;
    vpix = _vpix;

    h_one = make_float2(1.0,0.0);
    h_zero = make_float2(0.0,0.0);

    hipsparseStatus_t rv;
    rv = hipsparseCreate(&sparse);
    cusparse_check_rv("hipsparseCreate");
    rv = hipsparseCreateMatDescr(&descr);
    cusparse_check_rv("hipsparseCreateMatDescr");

    cell = 80.0; // 80 wavelengths == ~42' FoV

    allocate();
}

void Grid::allocate() {
    u.resize(nbl);
    v.resize(nbl);
    freq.resize(nchan);

    G_vals.resize(ncol());
    G_rows.resize(nrow()+1);
    G_cols.resize(ncol());
    G_cols0.resize(ncol());
    G_chan.resize(ncol());
    G_pix.resize(ncol());

    shift.resize(nchan);
    conj.resize(nbl);
}

void Grid::set_uv(const std::vector<float> &_u, const std::vector<float> &_v) {
    if (_u.size()!=nbl || _v.size()!=nbl) {
        char msg[1024];
        sprintf(msg, "Grid::set_uv array size error (u=%d v=%d nbl=%d)",
                _u.size(), _v.size(), nbl);
        throw std::invalid_argument(msg);
    }
    for (int i=0; i<nbl; i++) {
        u[i] = _u[i];
        v[i] = _v[i];
    }
}

void Grid::set_freq(const std::vector<float> &_freq) {
    if (_freq.size()!=nchan) {
        char msg[1024];
        sprintf(msg, "Grid::set_freq array size error (freq=%d nchan=%d)",
                _freq.size(), nchan);
        throw std::invalid_argument(msg);
    }
    for (int i=0; i<nchan; i++) { freq[i] = _freq[i]; }
}

void Grid::set_shift(const std::vector<int> &_shift) {
    if (_shift.size()!=nchan) {
        char msg[1024];
        sprintf(msg, "Grid::set_shift array size error (shift=%d nchan=%d)",
                _shift.size(), nchan);
        throw std::invalid_argument(msg);
    }
    maxshift=0;
    for (int i=0; i<nchan; i++) {
        if (_shift[i]>maxshift) { maxshift=_shift[i]; }
    }
    if (maxshift>ntime) { 
        char msg[1024];
        sprintf(msg, 
                "Grid::set_shift max shift out of range (maxshift=%d ntime=%d)",
                maxshift, ntime);
        throw std::invalid_argument(msg);
    }
    hipMemcpy(shift.d, _shift.data(), shift.size(), hipMemcpyHostToDevice);
}

void Grid::compute() {

    //printf("nrow=%d ncol=%d\n", nrow(), ncol());

    // compute grid pix location for each input vis
    nnz = 0;
    for (int ibl=0; ibl<nbl; ibl++) {
        for (int ichan=0; ichan<nchan; ichan++) {
            int x = round((u[ibl]*freq[ichan])/cell);
            int y = round((v[ibl]*freq[ichan])/cell); 
            if (y<0) { y*=-1; x*=-1; conj.h[ibl]=1; }
            else { conj.h[ibl]=0; }
            if (x<=upix/2 && x>=-upix/2 && y<vpix && y>=0) {
                if (x<0) x += upix;
                G_pix.h[nnz] = x*vpix + y;
                G_cols0.h[nnz] = ibl*nchan + ichan;
                nnz++;
            } 
        }
    }
    G_pix.h2d();
    G_cols0.h2d();
    conj.h2d();

    hipsparseStatus_t rv;

    // on GPU, sort and compress into CSR matrix format
    size_t pbuf_size;
    rv = hipsparseXcoosort_bufferSizeExt(sparse, nrow(), ncol(), nnz, 
            G_pix.d, G_cols.d, &pbuf_size);
    cusparse_check_rv("hipsparseXcoosort_bufferSizeExt");

    Array<char> pbuf(pbuf_size);
    Array<int> perm(nnz);
    rv = hipsparseCreateIdentityPermutation(sparse, nnz, perm.d);
    cusparse_check_rv("hipsparseCreateIdentityPermutation");

    rv = hipsparseXcoosortByRow(sparse, nrow(), ncol(), nnz,
            G_pix.d, G_cols0.d, perm.d, (void *)pbuf.d);
    cusparse_check_rv("hipsparseXcoosortByRow");

    rv = hipsparseXcoo2csr(sparse, G_pix.d, nnz, nrow(), G_rows.d,
            HIPSPARSE_INDEX_BASE_ZERO);
    cusparse_check_rv("hipsparseXcoo2csr");

    // Fill in normalization factors (number of vis per grid point)
    // TODO maybe these should just be set to 1.0?
    G_rows.d2h();
    for (int i=0; i<nrow(); i++) {
        for (int j=G_rows.h[i]; j<G_rows.h[i+1]; j++) {
            G_vals.h[j].x = 1.0/((float)G_rows.h[i+1] - (float)G_rows.h[i]);
            G_vals.h[j].y = 0.0;
        }
    }
    G_vals.h2d();

    // retrieve channel idx of each data point
    G_cols0.d2h();
    for (int i=0; i<nnz; i++) { G_chan.h[i] = G_cols0.h[i] % nchan; }
    G_chan.h2d();
}

// Call with nbl thread blocks
__global__ void conjugate_data(cdata *dat, int *conj, int nchan, int ntime) {
    const int ibl = blockIdx.x;
    const int offs = ibl*nchan*ntime;
    if (conj[ibl]) { 
        for (int i=threadIdx.x; i<nchan*ntime; i+=blockDim.x) {
            dat[offs+i].y *= -1.0;
        }
    }
}

void Grid::conjugate(Array<cdata,true> &data) {
    conjugate_data<<<nbl,512>>>(data.d, conj.d, nchan, ntime);
}

__global__ void adjust_cols(int *ocol, int *icol, int *chan,
        int *shift, int itime, int nchan, int nnz, int ntime) {
    const int ii = blockDim.x*blockIdx.x + threadIdx.x;
    __shared__ int lshift[2048]; // max nchan=2048 TODO 
    for (int i=threadIdx.x; i<nchan; i+=blockDim.x) {
        lshift[i] = shift[i];
    }
    __syncthreads();
    if (ii<nnz) { ocol[ii] = icol[ii]*ntime + lshift[chan[ii]] + itime; }
}

void Grid::operate(Array<cdata,true> &in, Array<cdata,true> &out, int itime) {
    if (in.len()!=nbl*nchan*ntime) {
        char msg[1024];
        sprintf(msg, "Grid::operate input array size (%d) != expected (%d)",
                in.len(), nbl*nchan*ntime);
        throw std::invalid_argument(msg);
    }
    if (out.len()!=upix*vpix) {
        char msg[1024];
        sprintf(msg, "Grid::operate output array size (%d) != expected (%d)",
                in.len(), upix*vpix);
        throw std::invalid_argument(msg);
    }
    operate(in.d, out.d, itime);
}

void Grid::operate(cdata *in, cdata *out, int itime) {
    if (itime>=ntime) {
        char msg[1024];
        sprintf(msg, "Grid::operate itime (%d) >= ntime (%d)", itime, ntime);
        throw std::invalid_argument(msg);
    }

    adjust_cols<<<nbl, nchan>>>(G_cols.d, G_cols0.d, G_chan.d, shift.d, 
            itime, nchan, nnz, ntime);

    hipsparseStatus_t rv;
    rv = hipsparseCcsrmv(sparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            nrow(), ncol()*ntime, nnz, &h_one, descr,
            G_vals.d, G_rows.d, G_cols.d,
            in, &h_zero, out);
    cusparse_check_rv("hipsparseCcsrmv");
}

